
// Kernel definition

#include <hip/hip_runtime.h>
#include "cmath"
#include <iostream>
#define N 10

__global__ void add(size_t n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (size_t i = index; i < n; i+=stride)
    {
        y[i] += x[i];
    }
    
}

int main() {
    float *A, *B;

    float maxError = 0.0f;

    hipMallocManaged(&A, N * sizeof(float));
    hipMallocManaged(&B, N * sizeof(float));
    for (size_t i = 0; i < N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    add<<<1, 1>>>(N, A, B);

    hipDeviceSynchronize();

    for (size_t i = 0; i < N; i++) {
        float error = 3.0f - B[i];
        maxError = fmax(maxError, error);
    }

    std::cout << "Max Error is " << maxError << std::endl;

    // Kernel invocation with N threads

    hipFree(A);
    hipFree(B);
}